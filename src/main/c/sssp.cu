#include <stdio.h>
#include <hip/hip_runtime.h>
#include <nvgraph.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <chrono>
#include <iostream>

void print_output(float *results, int nvertices);

void check(nvgraphStatus_t status) {
    if (status != NVGRAPH_STATUS_SUCCESS) {
        printf("ERROR : %d\n", status);
        exit(0);
    }
}

std::string getEpoch() {
    return std::to_string(std::chrono::duration_cast<std::chrono::milliseconds>
        (std::chrono::system_clock::now().time_since_epoch()).count());
}

// NVIDIA's SSSP implementation using nvGRAPH: https://docs.nvidia.com/cuda/nvgraph/index.html#nvgraph-sssp-example
int main(int argc, char **argv) {
    const size_t  n = 6, nnz = 10, vertex_numsets = 1, edge_numsets = 1;
    float *sssp_1_h;
    void** vertex_dim;

    // nvgraph variables
    nvgraphStatus_t status; nvgraphHandle_t handle;
    nvgraphGraphDescr_t graph;
    nvgraphCSCTopology32I_t CSC_input;
    hipDataType edge_dimT = HIP_R_32F;
    hipDataType* vertex_dimT;

    // Init host data
    sssp_1_h = (float*)malloc(n*sizeof(float));
    vertex_dim  = (void**)malloc(vertex_numsets*sizeof(void*));
    vertex_dimT = (hipDataType*)malloc(vertex_numsets*sizeof(hipDataType));
    CSC_input = (nvgraphCSCTopology32I_t) malloc(sizeof(struct nvgraphCSCTopology32I_st));
    vertex_dim[0]= (void*)sssp_1_h; vertex_dimT[0] = HIP_R_32F;
    float weights_h[] = {0.333333, 0.5, 0.333333, 0.5, 0.5, 1.0, 0.333333, 0.5, 0.5, 0.5};
    int destination_offsets_h[] = {0, 1, 3, 4, 6, 8, 10};
    int source_indices_h[] = {2, 0, 2, 0, 4, 5, 2, 3, 3, 4};

    check(nvgraphCreate(&handle));

    check(nvgraphCreateGraphDescr (handle, &graph));
    CSC_input->nvertices = n; CSC_input->nedges = nnz;
    CSC_input->destination_offsets = destination_offsets_h;
    CSC_input->source_indices = source_indices_h;

    // Set graph connectivity and properties (tranfers)
    check(nvgraphSetGraphStructure(handle, graph, (void*)CSC_input, NVGRAPH_CSC_32));
    check(nvgraphAllocateVertexData(handle, graph, vertex_numsets, vertex_dimT));
    check(nvgraphAllocateEdgeData  (handle, graph, edge_numsets, &edge_dimT));
    check(nvgraphSetEdgeData(handle, graph, (void*)weights_h, 0));

    std::chrono::steady_clock::time_point start = std::chrono::steady_clock::now();
    std::cout << "Processing starts at: " << getEpoch() << std::endl;

    // Solve
    int source_vert = 0;
    check(nvgraphSssp(handle, graph, 0,  &source_vert, 0));

    // Get and print result
    check(nvgraphGetVertexData(handle, graph, (void*)sssp_1_h, 0));
    std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
    std::cout << "Processing ends at: " << getEpoch() << std::endl;

    // Clean
    print_output(sssp_1_h, 6);
    free(sssp_1_h); free(vertex_dim);
    free(vertex_dimT); free(CSC_input);
    check(nvgraphDestroyGraphDescr(handle, graph));
    check(nvgraphDestroy(handle));

    return 0;
}

void print_output(float *results, int nvertices) {
    for (int i = 0; i < nvertices; i++) {
        printf("%f \n", results[i]);
    }
}